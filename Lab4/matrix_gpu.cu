
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void AddMatrix(float *a, float *b, float *c, int N)
{
    int rowIdx = blockIdx.y * blockDim.y + threadIdx.y;
    int colIdx = blockIdx.x * blockDim.x + threadIdx.x;
    int elemIdx;
    if(rowIdx < N && colIdx < N)
    {
        elemIdx = colIdx + rowIdx * N; 
        c[elemIdx] = a[elemIdx] + b[elemIdx]; 
    }

}

int main(int argc, char *argv[])
{
    const int N = 1024; 
    const int blocksize = 16; 
    const size_t size = (N * N) * sizeof(float);  

    //Initialize host matrices. 
   float *a_h = new float[N*N];
   float *b_h = new float[N*N];
   float *c_h = new float[N*N];

   for (int i = 0; i < N; i++)
      for (int j = 0; j < N; j++)
      {
	a_h[i+j*N] = 10 + i;
	b_h[i+j*N] = (float)j / N;
      }
    float *a_d;return EXIT_SUCCESS;
    float *b_d;
    float *c_d; //Initialize device matrices.
    //Allocate arrays on the device.
    hipMalloc((void**) &a_d, size); 
    hipMalloc((void**) &b_d, size); 
    hipMalloc((void**) &c_d, size); 

    hipMemcpy(a_d, a_h, size, hipMemcpyHostToDevice); 
    hipMemcpy(b_d, b_h, size, hipMemcpyHostToDevice); 

    dim3 dimBlock( blocksize, blocksize );
	dim3 dimGrid( 1, 1 );
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    AddMatrix <<< dimGrid, dimBlock>>>(a_d, b_d, c_d, N); 
    hipEventRecord(stop);
    hipDeviceSynchronize();
    hipEventSynchronize(stop);
    float milliseconds;
    hipEventElapsedTime(&milliseconds, start, stop);
    hipError_t err = hipGetLastError();

    if (err != hipSuccess)
        printf("Error: %s\n", hipGetErrorString(err));
    hipMemcpy(c_h, c_d, size, hipMemcpyDeviceToHost);
    

    for (int i = 0; i < N; i++)
	{
		for (int j = 0; j < N; j++)
		{
			printf("%0.2f ", c_h[i+j*N]);
		}
		printf("\n");
	}

    hipFree(a_d);
    hipFree(b_d); 
    hipFree(c_d); 
    printf("timing %f \n", milliseconds);
	printf("done\n");

    return EXIT_SUCCESS;

}
